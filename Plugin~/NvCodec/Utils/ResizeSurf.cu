#include "hip/hip_runtime.h"
#include "ResizeSurf.h"

#include <hip/hip_runtime.h>

__global__ void ResizeSurfNearestNeiborKernel(
    hipSurfaceObject_t srcSurface,
    int srcWidth,
    int srcHeight,
    hipSurfaceObject_t dstSurface,
    int dstWidth,
    int dstHeight,
    float scaleWidth,
    float scaleHeight)
{
    // calculate surface coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dstWidth || y >= dstHeight)
        return;

    uchar4 data;
    surf2Dread(&data, srcSurface, (x * scaleWidth) * 4, y * scaleHeight);

    // read from global memory and write to cuarray (via surface reference)
    surf2Dwrite(data, dstSurface, x * 4, y);
}

__global__ void ResizeSurfBilinearKernel(
    hipSurfaceObject_t srcSurface,
    int srcWidth,
    int srcHeight,
    hipSurfaceObject_t dstSurface,
    int dstWidth,
    int dstHeight,
    float scaleWidth,
    float scaleHeight)
{
    // calculate surface coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dstWidth || y >= dstHeight)
        return;

    int x0 = x * scaleWidth - 1;
    int x1 = x * scaleWidth + 1;
    int y0 = y * scaleHeight - 1;
    int y1 = y * scaleHeight + 1;

    if (x0 < 0) 
        x0 = 0;
    if (x1 >= srcWidth) 
        x1 = srcWidth - 1;
    if (y0 < 0) 
        y0 = 0;
    if (y1 >= srcHeight) 
        y1 = srcHeight - 1;

    uchar4 c00, c01, c10, c11;
    surf2Dread(&c00, srcSurface, x0 * 4, y0);
    surf2Dread(&c01, srcSurface, x0 * 4, y1);
    surf2Dread(&c10, srcSurface, x1 * 4, y0);
    surf2Dread(&c11, srcSurface, x1 * 4, y1);

    uchar4 data;
    data.x = 0.5f * (0.5f * c00.x + 0.5f * c01.x) + 0.5f * (0.5f * c10.x + 0.5f * c11.x);
    data.y = 0.5f * (0.5f * c00.y + 0.5f * c01.y) + 0.5f * (0.5f * c10.y + 0.5f * c11.y);
    data.z = 0.5f * (0.5f * c00.z + 0.5f * c01.z) + 0.5f * (0.5f * c10.z + 0.5f * c11.z);
    
    surf2Dwrite(data, dstSurface, x * 4, y);
}

hipError_t ResizeSurf(hipArray_t srcArray, hipArray_t dstArray)
{
    hipSurfaceObject_t srcSurface;
    HIP_RESOURCE_DESC srcResDesc;
    srcResDesc.flags = 0;
    srcResDesc.resType = HIP_RESOURCE_TYPE_ARRAY;
    srcResDesc.res.array.hArray = srcArray;

    hipError_t result = cuSurfObjectCreate(&srcSurface, &srcResDesc);
    if (result != hipSuccess)
        return result;

    hipSurfaceObject_t dstSurface;
    HIP_RESOURCE_DESC dstResDesc;
    dstResDesc.flags = 0;
    dstResDesc.resType = HIP_RESOURCE_TYPE_ARRAY;
    dstResDesc.res.array.hArray = dstArray;

    result = cuSurfObjectCreate(&dstSurface, &dstResDesc);
    if (result != hipSuccess)
        return result;

    HIP_ARRAY_DESCRIPTOR srcArrayDesc;
    result = hipArrayGetDescriptor(&srcArrayDesc, srcArray);
    if (result != hipSuccess)
        return result;

    HIP_ARRAY_DESCRIPTOR dstArrayDesc;
    result = hipArrayGetDescriptor(&dstArrayDesc, dstArray);
    if (result != hipSuccess)
        return result;

    int srcWidth = srcArrayDesc.Width;
    int srcHeight = srcArrayDesc.Height;
    int dstWidth = dstArrayDesc.Width;
    int dstHeight = dstArrayDesc.Height;

    dim3 dimBlock(8, 8, 1);

    int gridX = dstWidth / dimBlock.x + (dstWidth % dimBlock.x ? 1 : 0);
    int gridY = dstHeight / dimBlock.y + (dstHeight % dimBlock.y ? 1 : 0);
    dim3 dimGrid(gridX, gridY, 1);

    ResizeSurfBilinearKernel<<<dimGrid, dimBlock>>>(
        srcSurface,
        srcWidth,
        srcHeight,
        dstSurface,
        dstWidth,
        dstHeight,
        srcWidth / dstWidth,
        srcHeight / dstHeight);

    result = cuSurfObjectDestroy(srcSurface);
    if (result != hipSuccess)
        return result;
    return cuSurfObjectDestroy(dstSurface);
}
