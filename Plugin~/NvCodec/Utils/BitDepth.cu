/*
* Copyright 2017-2018 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

static __global__ void ConvertUInt8ToUInt16Kernel(uint8_t *dpUInt8, uint16_t *dpUInt16, int nSrcPitch, int nDestPitch, int nWidth, int nHeight)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x,
        y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= nWidth || y >= nHeight)
    {
        return;
    }
    int destStrideInPixels = nDestPitch / (sizeof(uint16_t));
    *(uchar2 *)&dpUInt16[y * destStrideInPixels + x] = uchar2{ 0, dpUInt8[y * nSrcPitch + x] };
}

static __global__ void ConvertUInt16ToUInt8Kernel(uint16_t *dpUInt16, uint8_t *dpUInt8, int nSrcPitch, int nDestPitch, int nWidth, int nHeight)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x,
        y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= nWidth || y >= nHeight)
    {
        return;
    }
    int srcStrideInPixels = nSrcPitch / (sizeof(uint16_t));
    dpUInt8[y * nDestPitch + x] = ((uchar2 *)&dpUInt16[y * srcStrideInPixels + x])->y;
}

void ConvertUInt8ToUInt16(uint8_t *dpUInt8, uint16_t *dpUInt16, int nSrcPitch, int nDestPitch, int nWidth, int nHeight)
{
    dim3 blockSize(16, 16, 1);
    dim3 gridSize(((uint32_t)nWidth + blockSize.x - 1) / blockSize.x, ((uint32_t)nHeight + blockSize.y - 1) / blockSize.y, 1);
    ConvertUInt8ToUInt16Kernel <<< gridSize, blockSize >>>(dpUInt8, dpUInt16, nSrcPitch, nDestPitch, nWidth, nHeight);
}

void ConvertUInt16ToUInt8(uint16_t *dpUInt16, uint8_t *dpUInt8, int nSrcPitch, int nDestPitch, int nWidth, int nHeight)
{
    dim3 blockSize(16, 16, 1);
    dim3 gridSize(((uint32_t)nWidth + blockSize.x - 1) / blockSize.x, ((uint32_t)nHeight + blockSize.y - 1) / blockSize.y, 1);
    ConvertUInt16ToUInt8Kernel <<<gridSize, blockSize >>>(dpUInt16, dpUInt8, nSrcPitch, nDestPitch, nWidth, nHeight);
}
